#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of 
 * students taking the CSC367H1 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited. 
 * All forms of distribution of this code, whether as given or with 
 * any changes, are expressly prohibited. 
 * 
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 * 
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2017 Bogdan Simion
 * -------------
*/

#include "kernels.h"

__global__ void kernel4(const int8_t *filter, int32_t dimension, 
        const int32_t *input, int32_t *output, int32_t width, int32_t height, int32_t *g_max, int32_t *g_min)
{
	int start = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ int32_t sdata_min[512];
	__shared__ int32_t sdata_max[512];
	unsigned int tid = threadIdx.x;
	sdata_min[tid] = 999999;
	sdata_max[tid] = -999999;


	int32_t min_v = 999999;
	int32_t max_v = -999999;
	// if (height % devProp.maxThreadsDim[0] > 0) {rows += 1;}
	for(int idx = start; idx < height * width; idx += gridDim.x * blockDim.x){
		int32_t sum = 0;
		// int initial_off = dimension / 2;
		int img_x = idx % width - dimension / 2;
		int img_y = idx / width - dimension / 2;
		
		for(int y = 0; y < dimension; y++){
			for(int x = 0; x < dimension; x ++){
				if(img_x >= 0 && img_x < width && img_y >= 0 && img_y < height){
					int fil_pos = dimension * y + x;
					int img_pos = width * img_y + img_x;
					sum += input[img_pos] * filter[fil_pos];
				}
				img_x++;
			}
			
			img_y++;
			img_x = idx % width - dimension / 2;
		}
		output[idx] = sum;
		if(sum > max_v) {max_v = sum;}
		if(sum < min_v) {min_v = sum;}	
		
	}
	sdata_min[tid] = min_v;
	sdata_max[tid] = max_v;
	__syncthreads();
	for (unsigned int s = blockDim.x/2; s > 32; s >>= 1) { 
	    if (tid < s) {
	    	if(sdata_max[tid] < sdata_max[tid + s]){
	    		sdata_max[tid] = sdata_max[tid + s];
	    	}
	    	if(sdata_min[tid + s] < sdata_min[tid]){
	    		sdata_min[tid] = sdata_min[tid + s];
	    	}
	    }
	    __syncthreads();
	}
	unsigned int blockSize = blockDim.x;
	if (tid < 32) {
		volatile int32_t* smem_max = sdata_max;
		volatile int32_t* smem_min = sdata_min;
		if (blockSize >= 64) {
			if(smem_max[tid] < smem_max[tid + 32]){
	    		smem_max[tid] = smem_max[tid + 32];
	    	}
	    	if(smem_min[tid + 32] < smem_min[tid]){
	    		smem_min[tid] = smem_min[tid + 32];
	    	}
    	}
		if (blockSize >= 32) {
			if(smem_max[tid] < smem_max[tid + 16]){
	    		smem_max[tid] = smem_max[tid + 16];
	    	}
	    	if(smem_min[tid + 16] < smem_min[tid]){
	    		smem_min[tid] = smem_min[tid + 16];
	    	}
    	}
		if (blockSize >= 16) {
			if(smem_max[tid] < smem_max[tid + 8]){
	    		smem_max[tid] = smem_max[tid + 8];
	    	}
	    	if(smem_min[tid + 8] < smem_min[tid]){
	    		smem_min[tid] = smem_min[tid + 8];
	    	}
    	}
		if (blockSize >=  8) {
			if(smem_max[tid] < smem_max[tid + 4]){
	    		smem_max[tid] = smem_max[tid + 4];
	    	}
	    	if(smem_min[tid + 4] < smem_min[tid]){
	    		smem_min[tid] = smem_min[tid + 4];
	    	}
    	}
		if (blockSize >=  4) {
			if(smem_max[tid] < smem_max[tid + 2]){
	    		smem_max[tid] = smem_max[tid + 2];
	    	}
	    	if(smem_min[tid + 2] < smem_min[tid]){
	    		smem_min[tid] = smem_min[tid + 2];
	    	}
    	}
		if (blockSize >=  2) {
			if(smem_max[tid] < smem_max[tid + 1]){
	    		smem_max[tid] = smem_max[tid + 1];
	    	}
	    	if(smem_min[tid + 1] < smem_min[tid]){
	    		smem_min[tid] = smem_min[tid + 1];
	    	}
    	}
	}
	if (tid == 0) { 
		g_max[blockIdx.x] = sdata_max[0]; 
		g_min[blockIdx.x] = sdata_min[0]; 

	}
}


__global__ void normalize4(int32_t *image, int32_t width, int32_t height,
        int32_t smallest, int32_t biggest)
{
	if(smallest != biggest){
		int start = threadIdx.x + blockIdx.x * blockDim.x;
		for(int idx = start; idx < height * width; idx += gridDim.x * blockDim.x){
			if (idx < width * height){
					image[idx] = ((image[idx] - smallest) * 255) / (biggest - smallest);
			}	
		
		}
	}
}
